#include "hip/hip_runtime.h"
#include "../header/cudaManager.h"

const unsigned int numCubesX = 20;
const unsigned int numCubesY = 20;
const unsigned int numCubesZ = 20;
const unsigned int numCubes  = numCubesX * numCubesY * numCubesZ;
const float cubeSize         = 1.0f;
const float cubeDistance     = cubeSize + 0.01f;
// restitution
const float e = 0.50;
// invMass=1/(densiti*vol)
const float invMass = 1.0f / (0.6f * cubeSize * cubeSize * cubeSize);
const float initVel = 1.70f;

__global__ void calculate_vel_and_pos(float4 *pos, float4 *vel) {
    // calculate index
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    // calculate uv coordinates
    float u = ((x * cubeDistance) - (cubeDistance * numCubesX / 2.0));
    float w = ((y * cubeDistance) - (cubeDistance * numCubesY / 2.0));
    float v = ((z * cubeDistance) - (cubeDistance * numCubesZ / 2.0));

    // write output vertex
    unsigned int i = x + numCubesY * (y + numCubesZ * z);
    pos[i] = make_float4(u, w, v, 1.0f);
    float lenght = sqrt(u * u + w * w + v * v);
    float u_vel  = (u / lenght) * initVel;
    float w_vel  = (w / lenght) * initVel;
    float v_vel  = (v / lenght) * initVel;
    vel[i] = make_float4(u_vel, w_vel, v_vel, 1.0f);
}


void initCubesDataKernal(float4 *ptr_pos, float4 *ptr_vel) {
    dim3 block(5, 5, 5);
    dim3 grid(numCubesX / block.x, numCubesY / block.y, numCubesZ / block.z);
    calculate_vel_and_pos << < grid, block >> > (ptr_pos, ptr_vel);
}



__global__ void calculate_update(float4 *pos, float4 *vel, float deltaTime, float bigCubeRad) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    float  dx = deltaTime * vel[i].x;
    float  dy = deltaTime * vel[i].y;
    float  dz = deltaTime * vel[i].z;
    float4 a  = pos[i];
    if ((a.x + dx > bigCubeRad - cubeSize) || (a.x + dx < -bigCubeRad)) {
        vel[i].x = -vel[i].x;
    }
    if ((a.y + dy > bigCubeRad - cubeSize) || (a.y + dy < -bigCubeRad)) {
        vel[i].y = -vel[i].y;
    }
    if ((a.z + dz > bigCubeRad - cubeSize) || (a.z + dz < -bigCubeRad)) {
        vel[i].z = -vel[i].z;
    }

    float u = pos[i].x + deltaTime * vel[i].x;
    float w = pos[i].y + deltaTime * vel[i].y;
    float v = pos[i].z + deltaTime * vel[i].z;
    pos[i] = make_float4(u, w, v, 1.0f);
}

__global__ void calculate_collision(float4 *pos, float4 *vel, float deltaTime) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < j) {
        float4 a = pos[i];
        float4 b = pos[j];
        if (((a.x <= b.x + cubeSize) && (a.x + cubeSize >= b.x)) &&
            ((a.y <= b.y + cubeSize) && (a.y + cubeSize >= b.y)) &&
            ((a.z <= b.z + cubeSize) && (a.z + cubeSize >= b.z))) {
            float  directionX = pos[j].x - pos[i].x;
            float  directionY = pos[j].y - pos[i].y;
            float  directionZ = pos[j].z - pos[i].z;
            float  len        = sqrt(directionX * directionX + directionY * directionY + directionZ * directionZ);
            float4 n          = make_float4(directionX / len, directionY / len, directionZ / len, 1.0f);

            float4 rv = make_float4(vel[j].x - vel[i].x, vel[j].y - vel[i].y, vel[j].z - vel[i].z, 1.0f);
            // // Calculate relative velocity in terms of the normal direction
            float velAlongNormal = rv.x * n.x + rv.y * n.y + rv.z * n.z;
            //
            // // Do not resolve if velocities are separating
            if (velAlongNormal > 0) return;

            // Calculate impulse scalar
            float k = (-(1 + e) * velAlongNormal) / invMass;
            // Apply impulse
            float4 impulse = make_float4(k * n.x, k * n.y, k * n.z, 1.0f);
            vel[i] = make_float4(vel[i].x - impulse.x, vel[i].y - impulse.y, vel[i].z - impulse.z, 1.0f);
            vel[j] = make_float4(vel[j].x + impulse.x, vel[j].y + impulse.y, vel[j].z + impulse.z, 1.0f);
        }
    }
}


void cubesUpdate(float4 *ptr_pos, float4 *ptr_vel, float bigCubeRad, float deltaTime) {
    // unsigned int triangleNumberN = numCubes - 1;
    // diagonal + 1 / 2 = diag^2+diag / (2*diag) [+1 for ceiling]
    // unsigned int dimToScale = (triangleNumberN + 2) / 2;
    dim3 block(32, 32);
    dim3 grid(numCubes / block.x, numCubes / block.y);
    calculate_collision << < grid, block  >> > (ptr_pos, ptr_vel, deltaTime);
    calculate_update << < numCubes / 64, 64 >> > (ptr_pos, ptr_vel, deltaTime, bigCubeRad / 2.0f);
}
