#include "hip/hip_runtime.h"
#include "../header/cudaManager.h"

const unsigned int numCubesX = 8;
const unsigned int numCubesY = 8;
const unsigned int numCubesZ = 8;
const unsigned int numCubes  = numCubesX * numCubesY * numCubesZ;
const float cubeSize         = numCubesY + 0.3f;

__global__ void calculate_vel_and_pos(float4 *pos, float4 *vel) {
    // calculate index
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    // calculate uv coordinates
    float offSet = (float)(1.0 / 2.0 * cubeSize);
    float u      = ((x / (float)numCubesX) * cubeSize - offSet);
    float w      = ((y / (float)numCubesY) * cubeSize - offSet);
    float v      = ((z / (float)numCubesZ) * cubeSize - offSet);

    // write output vertex
    pos[x + numCubesY * (y + numCubesZ * z)] = make_float4(u, w, v, 1.0f);
    float lenght   = sqrt(u * u + w * w + v * v);
    float totalVel = 1.1f;
    float u_vel    = (u / lenght) * totalVel;
    float w_vel    = (w / lenght) * totalVel;
    float v_vel    = (v / lenght) * totalVel;
    vel[x + numCubesY * (y + numCubesZ * z)] = make_float4(u_vel, w_vel, v_vel, 1.0f);
}


void initCubesDataKernal(float4 *ptr_pos, float4 *ptr_vel) {
    dim3 block(8, 8, 8);
    dim3 grid(numCubesX / block.x, numCubesY / block.y, numCubesZ / block.z);
    calculate_vel_and_pos << < grid, block >> > (ptr_pos, ptr_vel);
}



__global__ void calculate_update(float4 *pos, float4 *vel, float deltaTime, float bigCubeRad) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
    float4 oldPos  = pos[x + numCubesY * (y + numCubesZ * z)];
    if ((oldPos.x > bigCubeRad) || (oldPos.x < -bigCubeRad)) {
        vel[x + numCubesY * (y + numCubesZ * z)].x = -vel[x + numCubesY * (y + numCubesZ * z)].x;
    }
    if ((oldPos.y > bigCubeRad) || (oldPos.y < -bigCubeRad)) {
        vel[x + numCubesY * (y + numCubesZ * z)].y = -vel[x + numCubesY * (y + numCubesZ * z)].y;
    }
    if ((oldPos.z > bigCubeRad) || (oldPos.z < -bigCubeRad)) {
        vel[x + numCubesY * (y + numCubesZ * z)].z = -vel[x + numCubesY * (y + numCubesZ * z)].z;
    }


    float4 actualVel = vel[x + numCubesY * (y + numCubesZ * z)];
    float  u         = oldPos.x + deltaTime * actualVel.x;
    float  w         = oldPos.y + deltaTime * actualVel.y;
    float  v         = oldPos.z + deltaTime * actualVel.z;

    pos[x + numCubesY * (y + numCubesZ * z)] = make_float4(u, w, v, 1.0f);
}

void cubesUpdate(float4 *ptr_pos, float4 *ptr_vel, float bigCubeRad, float deltaTime) {
    dim3 block(8, 8, 8);
    dim3 grid(numCubesX / block.x, numCubesY / block.y, numCubesZ / block.z);
    calculate_update << < grid, block >> > (ptr_pos, ptr_vel, deltaTime, bigCubeRad / 2.0f);
}
